/*
################################################

Yigithan Yigit 2024

################################################
*/

#include <cugraph/algorithms.hpp>
#include <cugraph/graph.hpp>
#include <cugraph/utilities/high_res_timer.hpp>

#include <raft/core/handle.hpp>
#include <raft/util/cudart_utils.hpp>

#include <rmm/device_uvector.hpp>
#include <rmm/mr/device/cuda_memory_resource.hpp>

#include <algorithm>
#include <iterator>
#include <limits>
#include <numeric>
#include <vector>
#include <iostream>
#include <fstream>

#include "utils/test_graphs.hpp"
#include "utils/conversion_utilities.hpp"


int main(int argc, char** argv) {
    
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        std::cerr << "No CUDA devices found!" << std::endl;
        exit(1);
    }

    std::cout << "Available CUDA devices: " << deviceCount << std::endl;
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        std::cout << "Device " << i << ": " << prop.name << std::endl;
    }

    // Set default device
    hipError_t error = hipSetDevice(0);
    if (error != hipSuccess) {
        std::cerr << "hipSetDevice failed with error: " << hipGetErrorString(error) << std::endl;
        exit(EXIT_FAILURE);
    }

    //size_t stackSize = 1 << 24; // Set to 1 MB or appropriate size
    //hipDeviceSetLimit(hipLimitStackSize, stackSize);

    //hipStream_t stream{};
    //hipStreamCreate(&stream);

    // Device Handle/Context
    //raft::handle_t handle{stream};
    raft::handle_t handle{};
    HighResTimer hr_timer{};

    if (argc < 2) {
        std::cerr << std::endl
        << "Usage: " << argv[0] << " <path to mtx file>" << std::endl
        //<< "Max Level: " 
        << "Example: " << argv[0] << " ../../datasets/karate.mtx" << std::endl;
        exit(1);
    } else {
        std::cout << "Reading graph from file: " << argv[1] << std::endl;
    }

    std::string file_path = argv[1];

    // Check file_path is a valid file path
    std::ifstream
    file(file_path);
    if (!file.good()) {
        std::cerr << "Error: Invalid file path" << std::endl;
        exit(1);
    }

    auto usecase = cugraph::utilities::File_Usecase(file_path);

    RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
    hr_timer.start("Construct graph");

    auto [graph, edge_weights, d_renumber_map_labels] =
      cugraph::utilities::construct_graph<int64_t, int64_t, float, false, false>(
        handle, usecase, true, false);

    hr_timer.stop();
    hr_timer.display_and_clear(std::cout);

    auto graph_view = graph.view();
    auto edge_weight_view =
      edge_weights ? std::make_optional((*edge_weights).view()) : std::nullopt;

    RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
    hr_timer.start("Louvain");
    std::cout << "Running Louvain algorithm" << std::endl;

    try {
        std::cout << "clustering"<< std::endl;
        rmm::device_uvector<int64_t> clustering(
            graph_view.local_vertex_partition_range_size(), 
            handle.get_stream());

        std::cout << "Running Louvain" << std::endl;
        auto [num_levels, modularity] = cugraph::louvain(
            handle, 
            std::optional<std::reference_wrapper<raft::random::RngState>>{std::nullopt},
            graph.view(),
            edge_weight_view,
            clustering.data(),
            20,    // max_level
            // 1e-7f,  // threshold
            1e-2f,  // threshold
            1.0f    // resolution
        );


        std::cout << "Louvain Finished" << std::endl;
        RAFT_CUDA_TRY(hipDeviceSynchronize());
        hr_timer.stop();
        hr_timer.display_and_clear(std::cout);

        std::cout << "Number of levels: " << num_levels << std::endl;
        std::cout << "Modularity: " << modularity << std::endl;
        
    } catch (raft::exception const& e) {
        std::cerr << "Exception: " << e.what() << std::endl;
        return EXIT_FAILURE;
    } catch (thrust::system_error const& e) {
        std::cerr << "Thrust exception: " << e.what() << std::endl;
        return EXIT_FAILURE;
    } catch (std::exception const& e) {
        std::cerr << "Standard exception: " << e.what() << std::endl;
        return EXIT_FAILURE;
    }

    return 0;
}